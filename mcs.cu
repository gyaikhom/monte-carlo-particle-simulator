#include "hip/hip_runtime.h"
/*8:*/
#line 28 "./common.w"

__device__ double cuda_convert_radian_to_degree(double angle)
{
angle*= RADIAN_TO_DEGREE;
if(angle<0.0)angle+= 360.0;
return angle;
}

/*:8*//*14:*/
#line 46 "./vector.w"

__device__ void cuda_vector_zero(Vector v)
{
v[0]= v[1]= v[2]= 0.0;
v[3]= 1.0;
}

/*:14*//*16:*/
#line 67 "./vector.w"

__device__ void cuda_vector_homogenise(Vector v)
{
if(1.0==v[3])return;
if(0.0!=v[3]){
v[0]/= v[3];
v[1]/= v[3];
v[2]/= v[3];
v[3]= 1.0;
}
}


/*:16*//*18:*/
#line 93 "./vector.w"

__device__ void cuda_vector_copy(Vector u,const Vector v)
{
u[0]= v[0];
u[1]= v[1];
u[2]= v[2];
u[3]= v[3];
}


/*:18*//*20:*/
#line 113 "./vector.w"

__device__ double cuda_vector_magnitude(const Vector v)
{
return sqrt(v[0]*v[0]+
v[1]*v[1]+
v[2]*v[2]);
}


/*:20*//*22:*/
#line 144 "./vector.w"

__device__ void cuda_vector_normalize(const Vector v,Vector r)
{
double m= cuda_vector_magnitude(v);

if(m> 0.0){
r[0]= v[0]/m;
r[1]= v[1]/m;
r[2]= v[2]/m;
}else{
r[0]= 0.0;
r[1]= 0.0;
r[2]= 0.0;
}
}

/*:22*//*24:*/
#line 174 "./vector.w"

__device__ void cuda_vector_difference(const Vector u,const Vector v,Vector r)
{
r[0]= u[0]-v[0];
r[1]= u[1]-v[1];
r[2]= u[2]-v[2];
r[3]= 1.0;
}


/*:24*//*26:*/
#line 195 "./vector.w"

__device__ double cuda_vector_dot(const Vector u,const Vector v)
{
return u[0]*v[0]+u[1]*v[1]+u[2]*v[2];
}

/*:26*//*28:*/
#line 222 "./vector.w"

__device__ void cuda_vector_cross(const Vector u,const Vector v,Vector r)
{
r[0]= (u[1]*v[2]-u[2]*v[1]);
r[1]= (u[2]*v[0]-u[0]*v[2]);
r[2]= (u[0]*v[1]-u[1]*v[0]);
}


/*:28*//*30:*/
#line 250 "./vector.w"

__device__ double cuda_vector_angle_radian(const Vector u,const Vector v)
{
Vector a,b,c= ZERO_VECTOR;
double angle;
cuda_vector_normalize(u,a);
cuda_vector_normalize(v,b);
angle= acos(vector_dot(a,b));
cuda_vector_cross(u,v,c);
if(c[3]<0.0)return(TWICE_PI-angle);
return angle;
}


/*:30*//*32:*/
#line 274 "./vector.w"

__device__ double cuda_vector_angle_degree(const Vector u,const Vector v)
{
return RADIAN_TO_DEGREE*vector_angle_radian(u,v);
}


/*:32*//*34:*/
#line 294 "./vector.w"

__device__ double cuda_vector_distance(const Vector u,const Vector v)
{
double x,y,z;
x= u[0]-v[0];
y= u[1]-v[1];
z= u[2]-v[2];
return sqrt(x*x+y*y+z*z);
}
#line 1 "./matrix.w"


/*:34*//*219:*/
#line 2118 "./csg.w"

__device__ Containment cuda_is_inside_block(const Vector v,const Primitive*p)
{
if(v[0]<p->b.x0||v[0]> p->b.x1||v[1]<p->b.y0||v[1]
> p->b.y1||v[2]<p->b.z0||v[2]> p->b.z1)
return OUTSIDE;
if(v[0]==p->b.x0||v[0]==p->b.x1||v[1]==p->b.y0||v[1]
==p->b.y1||v[2]==p->b.z0||v[2]==p->b.z1)
return SURFACE;
return INSIDE;
}

/*:219*//*221:*/
#line 2161 "./csg.w"

__device__ Containment cuda_is_inside_sphere(const Vector v,const Primitive*p)
{
double delta= sqrt(v[0]*v[0]+v[1]*v[1]+v[2]*v[2]);
if(delta> p->s.radius)return OUTSIDE;
if(delta==p->s.radius)return SURFACE;
return INSIDE;
}


/*:221*//*225:*/
#line 2221 "./csg.w"

__device__ Containment cuda_is_inside_cylinder(const Vector v,const Primitive*p)
{
double delta;
if(v[1]<p->c.y0||v[1]> p->c.y1)return OUTSIDE;
/*226:*/
#line 2238 "./csg.w"

delta= sqrt(v[0]*v[0]+v[2]*v[2]);

/*:226*/
#line 2226 "./csg.w"
;
if(delta> p->c.radius)return OUTSIDE;
if(v[1]==p->c.y0||v[1]==p->c.y1||delta==
p->c.radius)
return SURFACE;
return INSIDE;
}

/*:225*//*239:*/
#line 2376 "./csg.w"

__device__ Containment cuda_is_inside_torus(const Vector v,const Primitive*p)
{
double gamma,gamma_deg,tau,tau_deg,delta,radial;
Vector tube_center,from_tube_center_to_v,temp;
/*232:*/
#line 2331 "./csg.w"

delta= sqrt(v[0]*v[0]+v[2]*v[2]);

/*:232*/
#line 2381 "./csg.w"
;
if(delta<p->t.r0||delta> p->t.r1)return OUTSIDE;

/*240:*/
#line 2392 "./csg.w"

cuda_vector_copy(temp,v);
temp[1]= 0.0;
gamma= cuda_vector_angle_radian(positive_xaxis_unit_vector,temp);
gamma_deg= cuda_convert_radian_to_degree(gamma);

/*:240*/
#line 2384 "./csg.w"
;
if(angle_outside_range(gamma_deg,p->t.phi_start,p->t.phi_end))
return OUTSIDE;
/*241:*/
#line 2398 "./csg.w"

/*235:*/
#line 2356 "./csg.w"

tube_center[0]= p->t.major*cos(gamma);
tube_center[1]= 0.0;
tube_center[2]= p->t.major*sin(gamma);

/*:235*/
#line 2399 "./csg.w"
;
/*242:*/
#line 2405 "./csg.w"

cuda_vector_difference(v,tube_center,from_tube_center_to_v);
radial= cuda_vector_magnitude(from_tube_center_to_v);

/*:242*/
#line 2400 "./csg.w"
;
if(radial> p->t.minor)return OUTSIDE;
/*243:*/
#line 2409 "./csg.w"

tau= cuda_vector_angle_radian(tube_center,from_tube_center_to_v);
tau_deg= cuda_convert_radian_to_degree(tau);

/*:243*/
#line 2402 "./csg.w"
;
if(angle_outside_range(tau,p->t.theta_start,p->t.theta_end))return OUTSIDE;

/*:241*/
#line 2387 "./csg.w"
;
/*238:*/
#line 2369 "./csg.w"

if(radial==p->t.minor)return SURFACE;
if(p->t.phi<360.0&&(gamma==p->t.phi_start||gamma==p->t.phi_end))
return SURFACE;
if(p->t.theta<360.0&&(tau==p->t.theta_start||tau==p->t.theta_end))
return SURFACE;

/*:238*/
#line 2388 "./csg.w"
;
return INSIDE;
}

/*:239*//*317:*/
#line 41 "./bstack.w"

#define MAX_BOOLEAN_STACK_SIZE 1024
typedef struct{
int tos,size;
bool v[MAX_BOOLEAN_STACK_SIZE];
}boolean_stack;
__device__ bool boolean_stack_init(boolean_stack*s)
{
if(NULL==s)return false;
s->tos= 0;
s->size= MAX_BOOLEAN_STACK_SIZE;
return true;
}
__device__ bool boolean_stack_push(boolean_stack*s,bool v)
{
if(s->tos==s->size)return false;
s->v[s->tos++]= v;
return true;
}
__device__ bool boolean_stack_pop(boolean_stack*s,bool*v)
{
if(0==s->tos)return false;
*v= s->v[--s->tos];
return true;
}

#line 1 "./postfix.w"


/*:317*//*320:*/
#line 29 "./postfix.w"

__device__ bool cuda_is_inside_primitive(Vector v,Primitive*p)
{
Containment c;
switch(p->type){
case BLOCK:c= cuda_is_inside_block(v,p);break;
case SPHERE:c= cuda_is_inside_sphere(v,p);break;
case CYLINDER:c= cuda_is_inside_cylinder(v,p);break;
case TORUS:c= cuda_is_inside_torus(v,p);break;
default:c= INVALID;
}
if(INSIDE==c||SURFACE==c)return true;
return false;
}

/*:320*/
